#include "hip/hip_runtime.h"
#include "pogs.h"

#include <algorithm>

#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>

#include "cml/cml_blas.cuh"
#include "cml/cml_vector.cuh"
#include "equil_helper.cuh"
#include "interface_defs.h"
#include "matrix/matrix.h"
#include "matrix/matrix_dense.h"
#include "matrix/matrix_sparse.h"
#include "projector/projector.h"
#include "projector/projector_direct.h"
#include "projector/projector_cgls.h"
#include "util.h"

#include "timer.h"

#define __HBAR__ \
"----------------------------------------------------------------------------\n"

namespace pogs {

template <typename T, typename M, typename P>
PogsImplementation<T, M, P>::PogsImplementation(const M &A)
    : _A(A), _P(_A),
      _de(0), _z(0), _zt(0),
      _rho(static_cast<T>(kRhoInit)),
      _done_init(false),
      _x(0), _y(0), _mu(0), _lambda(0), _optval(static_cast<T>(0.)),
      _final_iter(0),
      _abs_tol(static_cast<T>(kAbsTol)),
      _rel_tol(static_cast<T>(kRelTol)),
      _max_iter(kMaxIter),
      _init_iter(kInitIter),
      _verbose(kVerbose),
      _adaptive_rho(kAdaptiveRho),
      _gap_stop(kGapStop),
      _init_x(false), _init_lambda(false) {
  _x = new T[_A.Cols()]();
  _y = new T[_A.Rows()]();
  _mu = new T[_A.Cols()]();
  _lambda = new T[_A.Rows()]();
}

template <typename T, typename M, typename P>
int PogsImplementation<T, M, P>::_Init(const PogsObjective<T> *objective) {
  DEBUG_EXPECT(!_done_init);
  if (_done_init)
    return 1;
  _done_init = true;

  size_t m = _A.Rows();
  size_t n = _A.Cols();

  hipMalloc(&_de, (m + n) * sizeof(T));
  hipMalloc(&_z, (m + n) * sizeof(T));
  hipMalloc(&_zt, (m + n) * sizeof(T));
  hipMemset(_de, 0, (m + n) * sizeof(T));
  hipMemset(_z, 0, (m + n) * sizeof(T));
  hipMemset(_zt, 0, (m + n) * sizeof(T));
  CUDA_CHECK_ERR();

  hipblasHandle_t hdl;
  hipblasCreate(&hdl);
  CUDA_CHECK_ERR();

  _A.Init();
  _A.Equil(_de, _de + m,
           std::function<void(T*)>([objective](T *v){
               objective->constrain_d(v);
           }),
           std::function<void(T*)>([objective](T *v){
               objective->constrain_e(v);
           }));
  _nrmA = Norm2Est(hdl, &_A);
  CUDA_CHECK_ERR();

  _P.Init();
  CUDA_CHECK_ERR();

  hipblasDestroy(hdl);
  return 0;
}

template <typename T, typename M, typename P>
PogsStatus PogsImplementation<T, M, P>::Solve(PogsObjective<T> *objective) {
  double t0 = timer<double>();
  // Constants for adaptive-rho and over-relaxation.
  const T kDeltaMin       = static_cast<T>(1.05);
  const T kGamma          = static_cast<T>(1.01);
  const T kTau            = static_cast<T>(0.8);
  const T kAlpha          = static_cast<T>(1.7);
  const T kRhoMin         = static_cast<T>(1e-4);
  const T kRhoMax         = static_cast<T>(1e4);
  const T kKappa          = static_cast<T>(0.9);
  const T kOne            = static_cast<T>(1);
  const T kZero           = static_cast<T>(0);
  const T kProjTolMax     = static_cast<T>(1e-6);
  const T kProjTolMin     = static_cast<T>(1e-2);
  const T kProjTolPow     = static_cast<T>(2);
  const T kProjTolIni     = static_cast<T>(1e-5);
  const bool kUseExactTol = false;

  // Initialize Projector P and Matrix A.
  if (!_done_init)
    _Init(objective);

  // Extract values from pogs_data
  size_t m = _A.Rows();
  size_t n = _A.Cols();

  // Create cuBLAS handle.
  hipblasHandle_t hdl;
  hipblasCreate(&hdl);
  CUDA_CHECK_ERR();

  // Allocate data for ADMM variables.
  cml::vector<T> de    = cml::vector_view_array(_de, m + n);
  cml::vector<T> z     = cml::vector_view_array(_z, m + n);
  cml::vector<T> zt    = cml::vector_view_array(_zt, m + n);
  cml::vector<T> zprev = cml::vector_calloc<T>(m + n);
  cml::vector<T> ztemp = cml::vector_calloc<T>(m + n);
  cml::vector<T> z12   = cml::vector_calloc<T>(m + n);
  CUDA_CHECK_ERR();

  // Create views for x and y components.
  cml::vector<T> d     = cml::vector_subvector(&de, 0, m);
  cml::vector<T> e     = cml::vector_subvector(&de, m, n);
  cml::vector<T> x     = cml::vector_subvector(&z, 0, n);
  cml::vector<T> y     = cml::vector_subvector(&z, n, m);
  cml::vector<T> x12   = cml::vector_subvector(&z12, 0, n);
  cml::vector<T> y12   = cml::vector_subvector(&z12, n, m);
  cml::vector<T> xprev = cml::vector_subvector(&zprev, 0, n);
  cml::vector<T> yprev = cml::vector_subvector(&zprev, n, m);
  cml::vector<T> xtemp = cml::vector_subvector(&ztemp, 0, n);
  cml::vector<T> ytemp = cml::vector_subvector(&ztemp, n, m);
  CUDA_CHECK_ERR();

  // Scale objective to account for diagonal scaling e and d.
  objective->scale(d.data, e.data);
  CUDA_CHECK_ERR();

  // Initialize (x, lambda) from (x0, lambda0).
  if (_init_x) {
    cml::vector_memcpy(&xtemp, _x);
    cml::vector_div(&xtemp, &e);
    _A.Mul('n', kOne, xtemp.data, kZero, ytemp.data);
    hipDeviceSynchronize();
    cml::vector_memcpy(&z, &ztemp);
    CUDA_CHECK_ERR();
  }
  if (_init_lambda) {
    cml::vector_memcpy(&ytemp, _lambda);
    cml::vector_div(&ytemp, &d);
    _A.Mul('t', -kOne, ytemp.data, kZero, xtemp.data);
    hipDeviceSynchronize();
    cml::blas_scal(hdl, -kOne / _rho, &ztemp);
    cml::vector_memcpy(&zt, &ztemp);
    CUDA_CHECK_ERR();
  }

  // Make an initial guess for (x0 or lambda0).
  if (_init_x && !_init_lambda) {
    // Alternating projections to satisfy
    //   1. \lambda \in \partial f(y), \mu \in \partial g(x)
    //   2. \mu = -A^T\lambda
    cml::vector_set_all(&zprev, kZero);
    for (unsigned int i = 0; i < kInitIter; ++i) {
//      ProjSubgradEval(g_gpu, xprev.data, x.data, xtemp.data);
//      ProjSubgradEval(f_gpu, yprev.data, y.data, ytemp.data);
      _P.Project(xtemp.data, ytemp.data, kOne, xprev.data, yprev.data,
          kProjTolIni);
      hipDeviceSynchronize();
      CUDA_CHECK_ERR();
      cml::blas_axpy(hdl, -kOne, &ztemp, &zprev);
      cml::blas_scal(hdl, -kOne, &zprev);
    }
    // xt = -1 / \rho * \mu, yt = -1 / \rho * \lambda.
    cml::vector_memcpy(&zt, &zprev);
    cml::blas_scal(hdl, -kOne / _rho, &zt);
  } else if (_init_lambda && !_init_x) {
    ASSERT(false);
  }
  _init_x = _init_lambda = false;

  // Save initialization time.
  double time_init = timer<double>() - t0;

  // Signal start of execution.
  if (_verbose > 0) {
    Printf(__HBAR__
        "           POGS v%s - Proximal Graph Solver (GPU)                \n"
        "           (c) Christopher Fougner, Stanford University 2014-2015\n",
        POGS_VERSION);
  }
  if (_verbose > 1) {
    Printf(__HBAR__
        " Iter | pri res | pri tol | dua res | dua tol |   gap   | eps gap |"
        " pri obj\n" __HBAR__);
  }

  // Initialize scalars.
  T sqrtn_atol = std::sqrt(static_cast<T>(n)) * _abs_tol;
  T sqrtm_atol = std::sqrt(static_cast<T>(m)) * _abs_tol;
  T sqrtmn_atol = std::sqrt(static_cast<T>(m + n)) * _abs_tol;
  T delta = kDeltaMin, xi = static_cast<T>(1.0);
  unsigned int k = 0u, kd = 0u, ku = 0u;
  bool converged = false;
  T nrm_r, nrm_s, gap, eps_gap, eps_pri, eps_dua;

  for (;; ++k) {
    cml::vector_memcpy(&zprev, &z);

    // Evaluate Proximal Operators
    cml::blas_axpy(hdl, -kOne, &zt, &z);
    objective->prox(x.data, y.data, x12.data, y12.data, _rho);
    CUDA_CHECK_ERR();

    // Compute gap, optval, and tolerances.
    cml::blas_axpy(hdl, -kOne, &z12, &z);
    cml::blas_dot(hdl, &z, &z12, &gap);
    gap = std::abs(gap);
    eps_gap = sqrtmn_atol + _rel_tol * cml::blas_nrm2(hdl, &z) *
        cml::blas_nrm2(hdl, &z12);
    eps_pri = sqrtm_atol + _rel_tol * cml::blas_nrm2(hdl, &y12);
    eps_dua = _rho * (sqrtn_atol + _rel_tol * cml::blas_nrm2(hdl, &x));
    CUDA_CHECK_ERR();

    // Apply over relaxation.
    cml::vector_memcpy(&ztemp, &zt);
    cml::blas_axpy(hdl, kAlpha, &z12, &ztemp);
    cml::blas_axpy(hdl, kOne - kAlpha, &zprev, &ztemp);
    CUDA_CHECK_ERR();

    // Project onto y = Ax.
    T proj_tol = kProjTolMin / std::pow(static_cast<T>(k + 1), kProjTolPow);
    proj_tol = std::max(proj_tol, kProjTolMax);
    _P.Project(xtemp.data, ytemp.data, kOne, x.data, y.data, proj_tol);
    hipDeviceSynchronize();
    CUDA_CHECK_ERR();

    // Calculate residuals.
    cml::vector_memcpy(&ztemp, &zprev);
    cml::blas_axpy(hdl, -kOne, &z, &ztemp);
    hipDeviceSynchronize();
    nrm_s = _rho * (_nrmA * cml::blas_nrm2(hdl, &ytemp) +
        cml::blas_nrm2(hdl, &xtemp));

    cml::vector_memcpy(&ztemp, &z12);
    cml::blas_axpy(hdl, -kOne, &z, &ztemp);
    hipDeviceSynchronize();
    nrm_r = _nrmA * cml::blas_nrm2(hdl, &xtemp) + cml::blas_nrm2(hdl, &ytemp);

    // Calculate exact residuals only if necessary.
    bool exact = false;
    if ((nrm_r < 10 * eps_pri && nrm_s < 10 * eps_dua) || kUseExactTol) {
      cml::vector_memcpy(&ztemp, &z12);
      _A.Mul('n', kOne, x12.data, -kOne, ytemp.data);
      hipDeviceSynchronize();
      nrm_r = cml::blas_nrm2(hdl, &ytemp);
      cml::vector_memcpy(&ztemp, &z12);
      cml::blas_axpy(hdl, kOne, &zt, &ztemp);
      cml::blas_axpy(hdl, -kOne, &zprev, &ztemp);
      _A.Mul('t', kOne, ytemp.data, kOne, xtemp.data);
      hipDeviceSynchronize();
      nrm_s = _rho * cml::blas_nrm2(hdl, &xtemp);
      exact = true;
    }
    CUDA_CHECK_ERR();

    // Evaluate stopping criteria.
    converged = exact && nrm_r < eps_pri && nrm_s < eps_dua &&
        (!_gap_stop || gap < eps_gap);
    if (_verbose > 2 && k % 10  == 0 ||
        _verbose > 1 && k % 100 == 0 ||
        _verbose > 1 && converged) {
      T optval = objective->evaluate(x12.data, y12.data);
      Printf("%5d : %.2e  %.2e  %.2e  %.2e  %.2e  %.2e % .2e\n",
          k, nrm_r, eps_pri, nrm_s, eps_dua, gap, eps_gap, optval);
    }

    // Break if converged or there are nans
    if (converged || k == _max_iter - 1){ // || cml::vector_any_isnan(&zt))
      _final_iter = k;
      break;
    }

    // Update dual variable.
    cml::blas_axpy(hdl, kAlpha, &z12, &zt);
    cml::blas_axpy(hdl, kOne - kAlpha, &zprev, &zt);
    cml::blas_axpy(hdl, -kOne, &z, &zt);
    CUDA_CHECK_ERR();

    // Rescale rho.
    if (_adaptive_rho) {
      if (nrm_s < xi * eps_dua && nrm_r > xi * eps_pri &&
          kTau * static_cast<T>(k) > static_cast<T>(kd)) {
        if (_rho < kRhoMax) {
          _rho *= delta;
          cml::blas_scal(hdl, 1 / delta, &zt);
          delta = kGamma * delta;
          ku = k;
          if (_verbose > 3)
            Printf("+ rho %e\n", _rho);
        }
      } else if (nrm_s > xi * eps_dua && nrm_r < xi * eps_pri &&
          kTau * static_cast<T>(k) > static_cast<T>(ku)) {
        if (_rho > kRhoMin) {
          _rho /= delta;
          cml::blas_scal(hdl, delta, &zt);
          delta = kGamma * delta;
          kd = k;
          if (_verbose > 3)
            Printf("- rho %e\n", _rho);
        }
      } else if (nrm_s < xi * eps_dua && nrm_r < xi * eps_pri) {
        xi *= kKappa;
      } else {
        delta = kDeltaMin;
      }
      CUDA_CHECK_ERR();
    }
  }

  // Get optimal value
  _optval = objective->evaluate(x12.data, y12.data);

  // Check status
  PogsStatus status;
  if (!converged && k == _max_iter - 1)
    status = POGS_MAX_ITER;
  else if (!converged && k < _max_iter - 1)
    status = POGS_NAN_FOUND;
  else
    status = POGS_SUCCESS;

  // Print summary
  if (_verbose > 0) {
    Printf(__HBAR__
        "Status: %s\n"
        "Timing: Total = %3.2e s, Init = %3.2e s\n"
        "Iter  : %u\n",
        PogsStatusString(status).c_str(), timer<double>() - t0, time_init, k);
    Printf(__HBAR__
        "Error Metrics:\n"
        "Pri: "
        "|Ax - y|    / (abs_tol sqrt(m)     / rel_tol + |y|)          = %.2e\n"
        "Dua: "
        "|A'l + u|   / (abs_tol sqrt(n)     / rel_tol + |u|)          = %.2e\n"
        "Gap: "
        "|x'u + y'l| / (abs_tol sqrt(m + n) / rel_tol + |x,u| |y,l|)  = %.2e\n"
        __HBAR__, _rel_tol * nrm_r / eps_pri, _rel_tol * nrm_s / eps_dua,
        _rel_tol * gap / eps_gap);
  }

  // Scale x, y, lambda and mu for output.
  cml::vector_memcpy(&ztemp, &zt);
  cml::blas_axpy(hdl, -kOne, &zprev, &ztemp);
  cml::blas_axpy(hdl, kOne, &z12, &ztemp);
  cml::blas_scal(hdl, -_rho, &ztemp);
  cml::vector_mul(&ytemp, &d);
  cml::vector_div(&xtemp, &e);

  cml::vector_div(&y12, &d);
  cml::vector_mul(&x12, &e);

  // Copy results to output.
  cml::vector_memcpy(_x, &x12);
  cml::vector_memcpy(_y, &y12);
  cml::vector_memcpy(_mu, &xtemp);
  cml::vector_memcpy(_lambda, &ytemp);

  // Store z.
  cml::vector_memcpy(&z, &zprev);

  // Free memory.
  cml::vector_free(&z12);
  cml::vector_free(&zprev);
  cml::vector_free(&ztemp);
  hipblasDestroy(hdl);
  CUDA_CHECK_ERR();

  return status;
}

template <typename T, typename M, typename P>
PogsImplementation<T, M, P>::~PogsImplementation() {
  hipFree(_de);
  hipFree(_z);
  hipFree(_zt);
  _de = _z = _zt = 0;
  CUDA_CHECK_ERR();

  delete [] _x;
  delete [] _y;
  delete [] _mu;
  delete [] _lambda;
  _x = _y = _mu = _lambda = 0;
}

// Pogs for separable problems
namespace {
template <typename T>
struct Divide {
  __host__ __device__ FunctionObj<T> operator()(FunctionObj<T> &fi, T di) {
    fi.a /= di; fi.d /= di; fi.e /= di * di; return fi;
  }
};

template <typename T>
struct Multiply {
  __host__ __device__ FunctionObj<T> operator()(FunctionObj<T> &gi, T ei) {
    gi.a *= ei; gi.d *= ei; gi.e *= ei * ei; return gi;
  }
};

template <typename T>
class PogsObjectiveSeparable : public PogsObjective<T> {
 private:
   thrust::device_vector<FunctionObj<T> > f, g;
 public:
  PogsObjectiveSeparable(const std::vector<FunctionObj<T> >& f,
                         const std::vector<FunctionObj<T> >& g)
      : f(f), g(g) { }

  T evaluate(const T *x, const T *y) const {
    return FuncEval(f, y) + FuncEval(g, x);
  }
  void prox(const T *x_in, const T *y_in, T *x_out, T *y_out, T rho) const {
    ProxEval(g, rho, x_in, x_out);
    ProxEval(f, rho, y_in, y_out);
  }

  void scale(const T *d, const T *e) {
    thrust::transform(f.begin(), f.end(), thrust::device_pointer_cast(d),
        f.begin(), Divide<T>());
    thrust::transform(g.begin(), g.end(), thrust::device_pointer_cast(e),
        g.begin(), Multiply<T>());
  }

  void constrain_d(T *d) const { }
  void constrain_e(T *e) const { }
};
}  // namespace

// Implementation of PogsSeparable
template <typename T, typename M, typename P>
PogsSeparable<T, M, P>::PogsSeparable(const M& A)
    : PogsImplementation<T, M, P>(A) { }

template <typename T, typename M, typename P>
PogsSeparable<T, M, P>::~PogsSeparable() { }

template <typename T, typename M, typename P>
PogsStatus PogsSeparable<T, M, P>::Solve(const std::vector<FunctionObj<T>>& f,
                                         const std::vector<FunctionObj<T>>& g) {
  PogsObjectiveSeparable<T> pogs_obj(f, g);
  return this->PogsImplementation<T, M, P>::Solve(&pogs_obj);
}

// Pogs for cone problems
namespace {

template <typename T>
struct Square : thrust::unary_function<T, T> {
  inline __host__ __device__ T operator()(const T &x) const {
    return x * x;
  }
};

template <typename T>
struct Updater {
  T rho;
  Updater(T rho) : rho(rho) { }
  __host__ __device__ T operator()(T ci, T xi) { return xi - ci / rho; };
};

template <typename T>
class PogsObjectiveCone : public PogsObjective<T> {
 private:
  T c_scale;
  thrust::device_vector<T> b, c;
  const std::vector<ConeConstraintRaw> &Kx, &Ky;
  std::vector<hipStream_t> streams_x, streams_y;

 public:
  PogsObjectiveCone(const std::vector<T>& b,
                    const std::vector<T>& c,
                    const std::vector<ConeConstraintRaw>& Kx,
                    const std::vector<ConeConstraintRaw>& Ky)
      : b(b), c(c), Kx(Kx), Ky(Ky) {
    streams_x.resize(Kx.size());
    streams_y.resize(Ky.size());
    for (auto &stream : streams_x) {
      hipStreamCreate(&stream);
    }
    for (auto &stream : streams_y) {
      hipStreamCreate(&stream);
    }
  }

  ~PogsObjectiveCone() {
    for (auto &stream : streams_x) {
      hipStreamDestroy(stream);
    }
    for (auto &stream : streams_y) {
      hipStreamDestroy(stream);
    }
  }

  T evaluate(const T *x, const T*) const {
    return thrust::inner_product(c.begin(), c.end(),
        thrust::device_pointer_cast(x), static_cast<T>(0)) / c_scale;
  }

  void prox(const T *x_in, const T *y_in, T *x_out, T *y_out, T rho) const {
    hipMemcpy(x_out, x_in, c.size() * sizeof(T), hipMemcpyDeviceToDevice);
    thrust::transform(c.begin(), c.end(), thrust::device_pointer_cast(x_out),
        thrust::device_pointer_cast(x_out), Updater<T>(rho));

    hipMemcpy(y_out, y_in, b.size() * sizeof(T), hipMemcpyDeviceToDevice);
    thrust::transform(b.begin(), b.end(), thrust::device_pointer_cast(y_out),
        thrust::device_pointer_cast(y_out), thrust::minus<T>());
    hipDeviceSynchronize();
    CUDA_CHECK_ERR();

    ProxEvalConeGpu(Kx, streams_x, c.size(), x_out, x_out);
    hipDeviceSynchronize();
    CUDA_CHECK_ERR();
    ProxEvalConeGpu(Ky, streams_y, b.size(), y_out, y_out);
    hipDeviceSynchronize();
    CUDA_CHECK_ERR();

    thrust::transform(b.begin(), b.end(), thrust::device_pointer_cast(y_out),
        thrust::device_pointer_cast(y_out), thrust::minus<T>());
  }

  void scale(const T *d, const T *e) {
    thrust::transform(c.begin(), c.end(), thrust::device_pointer_cast(e),
        c.begin(), thrust::multiplies<T>());
    thrust::transform(b.begin(), b.end(), thrust::device_pointer_cast(d),
        b.begin(), thrust::multiplies<T>());

    c_scale = 1 / std::sqrt(thrust::transform_reduce(c.begin(), c.end(),
        Square<T>(), static_cast<T>(0), thrust::plus<T>()));

    thrust::transform(c.begin(), c.end(),
        thrust::constant_iterator<T>(c_scale), c.begin(),
        thrust::multiplies<T>());
  }

  // Average the e_i in Kx
  void constrain_e(T *e) const {
    for (auto& cone : Kx) {
      if (IsSeparable(cone.cone))
        continue;

      using ValIt = thrust::device_ptr<T>;
      using IdxIt = thrust::device_ptr<CONE_IDX>;

      thrust::permutation_iterator<ValIt, IdxIt> iter(
          thrust::device_pointer_cast(e),
          thrust::device_pointer_cast(cone.idx));
      T sum = thrust::reduce(iter, iter + cone.size);
      thrust::fill(iter, iter + cone.size, sum / cone.size);
      CUDA_CHECK_ERR();
    }
  }

  // Average the d_i in Ky
  void constrain_d(T *d) const {
    for (auto& cone : Ky) {
      if (IsSeparable(cone.cone))
        continue;

      using ValIt = thrust::device_ptr<T>;
      using IdxIt = thrust::device_ptr<CONE_IDX>;

      thrust::permutation_iterator<ValIt, IdxIt> iter(
          thrust::device_pointer_cast(d),
          thrust::device_pointer_cast(cone.idx));
      T sum = thrust::reduce(iter, iter + cone.size);
      thrust::fill(iter, iter + cone.size, sum / cone.size);
      CUDA_CHECK_ERR();
    }
  }
};

void MakeRawCone(const std::vector<ConeConstraint> &K,
                 std::vector<ConeConstraintRaw> *K_raw) {
  for (const auto& cone_constraint : K) {
    ConeConstraintRaw raw;
    raw.size = cone_constraint.idx.size();
    hipMalloc(&(raw.idx), raw.size * sizeof(CONE_IDX));
    hipMemcpy(raw.idx, cone_constraint.idx.data(), raw.size * sizeof(CONE_IDX),
        hipMemcpyHostToDevice);
    CUDA_CHECK_ERR();
    raw.cone = cone_constraint.cone;
    K_raw->push_back(raw);
  }
}

}  // namespace

// Implementation of PogsCone
template <typename T, typename M, typename P>
PogsCone<T, M, P>::PogsCone(const M& A,
                            const std::vector<ConeConstraint>& Kx,
                            const std::vector<ConeConstraint>& Ky)
    : PogsImplementation<T, M, P>(A) , valid_cones(true) {
  valid_cones = ValidCone(Kx, A.Cols()) && ValidCone(Ky, A.Rows());
  MakeRawCone(Kx, &this->Kx);
  MakeRawCone(Ky, &this->Ky);
}

template <typename T, typename M, typename P>
PogsCone<T, M, P>::~PogsCone() {
  for (const auto& cone_constraint : this->Kx)
    hipFree(cone_constraint.idx);
  for (const auto& cone_constraint : this->Ky)
    hipFree(cone_constraint.idx);
}

template <typename T, typename M, typename P>
PogsStatus PogsCone<T, M, P>::Solve(const std::vector<T>& b,
                                    const std::vector<T>& c) {
  if (!valid_cones)
    return POGS_INVALID_CONE;
  PogsObjectiveCone<T> pogs_obj(b, c, Kx, Ky);
  return this->PogsImplementation<T, M, P>::Solve(&pogs_obj);
}

// Explicit template instantiation.
#if !defined(POGS_DOUBLE) || POGS_DOUBLE==1
// Dense direct.
template class PogsSeparable<double, MatrixDense<double>,
    ProjectorDirect<double, MatrixDense<double> > >;
template class PogsSeparable<double, MatrixDense<double>,
    ProjectorCgls<double, MatrixDense<double> > >;
template class PogsSeparable<double, MatrixSparse<double>,
    ProjectorCgls<double, MatrixSparse<double> > >;

template class PogsCone<double, MatrixDense<double>,
    ProjectorDirect<double, MatrixDense<double> > >;
template class PogsCone<double, MatrixDense<double>,
    ProjectorCgls<double, MatrixDense<double> > >;
template class PogsCone<double, MatrixSparse<double>,
    ProjectorCgls<double, MatrixSparse<double> > >;
#endif

#if !defined(POGS_SINGLE) || POGS_SINGLE==1
template class PogsSeparable<float, MatrixDense<float>,
    ProjectorDirect<float, MatrixDense<float> > >;
template class PogsSeparable<float, MatrixDense<float>,
    ProjectorCgls<float, MatrixDense<float> > >;
template class PogsSeparable<float, MatrixSparse<float>,
    ProjectorCgls<float, MatrixSparse<float> > >;

template class PogsCone<float, MatrixDense<float>,
    ProjectorDirect<float, MatrixDense<float> > >;
template class PogsCone<float, MatrixDense<float>,
    ProjectorCgls<float, MatrixDense<float> > >;
template class PogsCone<float, MatrixSparse<float>,
    ProjectorCgls<float, MatrixSparse<float> > >;
#endif

}  // namespace pogs

