#include "hip/hip_runtime.h"
#include <hipsparse.h>
#include <hipblas.h>

#include "sinkhorn_knopp.cuh"

#include <cmath>
#include <algorithm>
#include <vector>

#include "_interface_defs.h"
#include "cml/cml_linalg.cuh"
#include "cml/cml_spblas.cuh"
#include "cml/cml_spmat.cuh"
#include "cml/cml_vector.cuh"
#include "pogs.h"
//#include "timer.hpp"

// Apply operator to h.a and h.d.
template <typename T, typename Op>
struct ApplyOp: thrust::binary_function<FunctionObj<T>, FunctionObj<T>, T> {
  Op binary_op;
  ApplyOp(Op binary_op) : binary_op(binary_op) { }
  __device__ FunctionObj<T> operator()(FunctionObj<T> &h, T x) {
    h.a = binary_op(h.a, x); h.d = binary_op(h.d, x);
    return h;
  }
};

// Proximal Operator Graph Solver.
template<typename T, typename M>
int Pogs(PogsData<T, M> *pogs_data) {
  // Constants for adaptive-rho and over-relaxation.
  const T kDeltaMin = static_cast<T>(1.05);
  const T kGamma = static_cast<T>(1.01);
  const T kTau = static_cast<T>(0.8);
  const T kAlpha = static_cast<T>(1.7);
  const T kKappa = static_cast<T>(0.9);
  const T kOne = static_cast<T>(1);
  const T kZero = static_cast<T>(0);
  const T kTol = static_cast<T>(1e-3);
  const T kRhoMax = static_cast<T>(1e4);
  const T kRhoMin = static_cast<T>(1e-4);
  const CBLAS_ORDER kOrd = M::Ord == ROW ? CblasRowMajor : CblasColMajor;

  int err = 0;

  // Extract values from pogs_data
  int m = pogs_data->m, n = pogs_data->n, nnz = pogs_data->A.nnz;
  T rho = pogs_data->rho;
  thrust::device_vector<FunctionObj<T> > f = pogs_data->f;
  thrust::device_vector<FunctionObj<T> > g = pogs_data->g;
 
  // Create cuBLAS hdl.
  hipblasHandle_t d_hdl;
  hipblasCreate(&d_hdl);
  hipsparseHandle_t s_hdl;
  hipsparseCreate(&s_hdl);
  hipsparseMatDescr_t descr;
  hipsparseCreateMatDescr(&descr);

  // Allocate data for ADMM variables.
  bool pre_process = true;
  cml::vector<T> de, z, zt;
  cml::vector<T> zprev = cml::vector_calloc<T>(m + n);
  cml::vector<T> z12 = cml::vector_calloc<T>(m + n);
  cml::spmat<T, typename M::I_t, kOrd> A;
  if (pogs_data->factors.val != 0) {
    hipMemcpy(&rho, pogs_data->factors.val, sizeof(T), hipMemcpyDeviceToHost);
    pre_process = (rho == 0);
    if (pre_process)
      rho = pogs_data->rho;
    de = cml::vector_view_array(pogs_data->factors.val + 1, m + n);
    z = cml::vector_view_array(pogs_data->factors.val + 1 + m + n, m + n);
    zt = cml::vector_view_array(pogs_data->factors.val + 1 + 2 * (m + n),
        m + n);
    A = cml::spmat<T, typename M::I_t, kOrd>(
        pogs_data->factors.val + 1 + 3 * (m + n),
        pogs_data->factors.ind, pogs_data->factors.ptr, m, n,
        pogs_data->factors.nnz);
  } else {
    de = cml::vector_calloc<T>(m + n);
    z = cml::vector_calloc<T>(m + n);
    zt = cml::vector_calloc<T>(m + n);
    A = cml::spmat_alloc<T, typename M::I_t, kOrd>(m, n, nnz);
  }

  if (de.data == 0 || z.data == 0 || zt.data == 0 || zprev.data == 0 ||
      z12.data == 0 || A.val == 0 || A.ind == 0 || A.ptr == 0) {
    err = 1;
  }

  // Create views for x and y components.
  cml::vector<T> d = cml::vector_subvector(&de, 0, m);
  cml::vector<T> e = cml::vector_subvector(&de, m, n);
  cml::vector<T> x = cml::vector_subvector(&z, 0, n);
  cml::vector<T> y = cml::vector_subvector(&z, n, m);
  cml::vector<T> x12 = cml::vector_subvector(&z12, 0, n);
  cml::vector<T> y12 = cml::vector_subvector(&z12, n, m);

  if (pre_process && !err) {
    cml::spmat_memcpy(s_hdl, &A, pogs_data->A.val, pogs_data->A.ind,
        pogs_data->A.ptr);
    err = sinkhorn_knopp::Equilibrate(s_hdl, d_hdl, descr, &A, &d, &e);

    // if (!err) {
    //   // TODO: Issue warning if x == NULL or y == NULL
    //   // Initialize x and y from x0 or/and y0
    //   if (pogs_data->init_x && !pogs_data->init_y && pogs_data->x) {
    //     cml::vector_memcpy(&x, pogs_data->x);
    //     cml::vector_div(&x, &e);
    //     cml::spblas_gemv(s_hdl, HIPSPARSE_OPERATION_NON_TRANSPOSE, descr, kOne,
    //         &A, &x, kZero, &y);
    //   } else if (pogs_data->init_y && !pogs_data->init_x && pogs_data->y) {
    //     cml::vector_memcpy(&y, pogs_data->y);
    //     cml::vector_mul(&y, &d);
    //     cml::vector_set_all(&x, kZero);
    //     cml::spblas_solve(s_hdl, d_hdl, descr, &A, static_cast<T>(1e-4), &y, &x,
    //         static_cast<T>(1e-6), 100, true);
    //     cml::spblas_gemv(s_hdl, HIPSPARSE_OPERATION_NON_TRANSPOSE, descr, kOne,
    //         &A, &x, kZero, &y);
    //   } else if (pogs_data->init_x && pogs_data->init_y &&
    //       pogs_data->x && pogs_data->y) {
    //     cml::vector_memcpy(&y, pogs_data->y);
    //     cml::vector_mul(&y, &d);
    //     cml::vector_memcpy(&x, pogs_data->x);
    //     cml::vector_div(&x, &e);
    //     cml::vector_memcpy(&x12, &x);
    //     cml::spblas_gemv(s_hdl, HIPSPARSE_OPERATION_NON_TRANSPOSE, descr, -kOne,
    //         &A, &x, kOne, &y);
    //     cml::spblas_solve(s_hdl, d_hdl, descr, &A, kOne, &y, &x12,
    //         static_cast<T>(1e-6), 100, true);
    //     cml::blas_axpy(d_hdl, -kOne, &x12, &x);
    //     cml::spblas_gemv(s_hdl, HIPSPARSE_OPERATION_NON_TRANSPOSE, descr, kOne,
    //         &A, &x, kZero, &y);
    //   }
    // }
  }

  // Scale f and g to account for diagonal scaling e and d.
  if (!err) {
    thrust::transform(f.begin(), f.end(), thrust::device_pointer_cast(d.data),
        f.begin(), ApplyOp<T, thrust::divides<T> >(thrust::divides<T>()));
    thrust::transform(g.begin(), g.end(), thrust::device_pointer_cast(e.data),
        g.begin(), ApplyOp<T, thrust::multiplies<T> >(thrust::multiplies<T>()));
  }

  // Signal start of execution.
  if (!pogs_data->quiet)
    Printf("   #      res_pri    eps_pri   res_dual   eps_dual"
           "        gap    eps_gap  objective\n");

  // Initialize scalars.
  T sqrtn_atol = std::sqrt(static_cast<T>(n)) * pogs_data->abs_tol;
  T sqrtm_atol = std::sqrt(static_cast<T>(m)) * pogs_data->abs_tol;
  T delta = kDeltaMin, xi = static_cast<T>(1.0);
  unsigned int kd = 0, ku = 0;
  bool converged = false;

  //double t = timer<double>();
  for (unsigned int k = 0; !err; ++k) {
    cml::vector_memcpy(&zprev, &z);

    // Evaluate Proximal Operators
    cml::blas_axpy(d_hdl, -kOne, &zt, &z);
    ProxEval(g, rho, x.data, x.stride, x12.data, x12.stride);
    ProxEval(f, rho, y.data, y.stride, y12.data, y12.stride);

    // Compute dual variable.
    T nrm_r = 0, nrm_s = 0, gap;
    cml::blas_axpy(d_hdl, -kOne, &z12, &z);
    cml::blas_dot(d_hdl, &z, &z12, &gap);
    gap = std::abs(gap);
    pogs_data->optval = FuncEval(f, y12.data, 1) + FuncEval(g, x12.data, 1);
    T eps_gap = std::sqrt(static_cast<T>(m + n)) * pogs_data->abs_tol +
        pogs_data->rel_tol * cml::blas_nrm2(d_hdl, &z) *
        cml::blas_nrm2(d_hdl, &z12);
    T eps_pri = sqrtm_atol + pogs_data->rel_tol * cml::blas_nrm2(d_hdl, &z12);
    T eps_dua = sqrtn_atol + pogs_data->rel_tol * rho * 
        cml::blas_nrm2(d_hdl, &z);

    if (converged || k == pogs_data->max_iter)
      break;

    // Project and Update Dual Variables
    cml::vector_memcpy(&y, &y12);
    cml::spblas_gemv(s_hdl, HIPSPARSE_OPERATION_NON_TRANSPOSE, descr, -kOne, &A,
        &x12, kOne, &y);
    nrm_r = cml::blas_nrm2(d_hdl, &y);
    cml::vector_set_all(&x, kZero);
    cml::spblas_solve(s_hdl, d_hdl, descr, &A, kOne, &y, &x, kTol, 5, true);
    cml::blas_axpy(d_hdl, kOne, &x12, &x);
    cml::spblas_gemv(s_hdl, HIPSPARSE_OPERATION_NON_TRANSPOSE, descr, kOne, &A,
        &x, kZero, &y);

    // Apply over relaxation.
    cml::blas_scal(d_hdl, kAlpha, &z);
    cml::blas_axpy(d_hdl, kOne - kAlpha, &zprev, &z);

    // Update dual variable.
    cml::blas_axpy(d_hdl, kAlpha, &z12, &zt);
    cml::blas_axpy(d_hdl, kOne - kAlpha, &zprev, &zt);
    cml::blas_axpy(d_hdl, -kOne, &z, &zt);

    bool exact = false;
    cml::blas_axpy(d_hdl, -kOne, &zprev, &z12);
    cml::blas_axpy(d_hdl, -kOne, &z, &zprev);
    nrm_s = rho * cml::blas_nrm2(d_hdl, &zprev);
    if (nrm_r < eps_pri && nrm_s < eps_dua) {
      cml::spblas_gemv(s_hdl, HIPSPARSE_OPERATION_TRANSPOSE, descr, kOne, &A,
          &y12, kOne, &x12);
      nrm_s = rho * cml::blas_nrm2(d_hdl, &x12);
      exact = true;
    }

    // Evaluate stopping criteria.
    converged = exact && nrm_r < eps_pri && nrm_s < eps_dua &&
        (!pogs_data->gap_stop || gap < eps_gap);
    if (!pogs_data->quiet && (k % 10 == 0 || converged))
      Printf("%4d :  %.3e  %.3e  %.3e  %.3e  %.3e  %.3e  %.3e\n",
          k, nrm_r, eps_pri, nrm_s, eps_dua, gap, eps_gap, pogs_data->optval);

    // Rescale rho.
    if (pogs_data->adaptive_rho && !converged) {
      if (nrm_s < xi * eps_dua && nrm_r > xi * eps_pri &&
          kTau * static_cast<T>(k) > static_cast<T>(kd)) {
        if (rho < kRhoMax) {
          rho *= delta;
          cml::blas_scal(d_hdl, 1 / delta, &zt);
          delta = kGamma * delta;
          ku = k;
          Printf("+ rho %e\n", rho);
        }
      } else if (nrm_s > xi * eps_dua && nrm_r < xi * eps_pri &&
          kTau * static_cast<T>(k) > static_cast<T>(ku)) {
        if (rho > kRhoMin) {
          rho /= delta;
          cml::blas_scal(d_hdl, delta, &zt);
          delta = kGamma * delta;
          kd = k;
          Printf("- rho %e\n", rho);
        }
      } else if (nrm_s < xi * eps_dua && nrm_r < xi * eps_pri) {
        xi *= kKappa;
      } else {
        delta = std::max(delta / kGamma, kDeltaMin);
      }
    }
  }
  //Printf("TIME = %e\n", timer<double>() - t);

  // Scale x, y and l for output.
  cml::vector_div(&y12, &d);
  cml::vector_mul(&x12, &e);
  cml::vector_mul(&y, &d);
  cml::blas_scal(d_hdl, rho, &y);

  // Copy results to output.
  if (pogs_data->y != 0 && !err)
    cml::vector_memcpy(pogs_data->y, &y12);
  if (pogs_data->x != 0 && !err)
    cml::vector_memcpy(pogs_data->x, &x12);
  if (pogs_data->l != 0 && !err)
    cml::vector_memcpy(pogs_data->l, &y);

  // Store rho and free memory.
  if (pogs_data->factors.val != 0 && !err) {
    hipMemcpy(pogs_data->factors.val, &rho, sizeof(T), hipMemcpyHostToDevice);
    cml::vector_memcpy(&z, &zprev);
  } else {
    cml::vector_free(&de);
    cml::vector_free(&z);
    cml::vector_free(&zt);
    cml::spmat_free(&A);
  }
  cml::vector_free(&z12);
  cml::vector_free(&zprev);

  return err;
}

template <typename T, typename I, POGS_ORD O>
int AllocSparseFactors(PogsData<T, Sparse<T, I, O> > *pogs_data) {
  size_t m = pogs_data->m, n = pogs_data->n, nnz = pogs_data->A.nnz;
  size_t flen = 1 + 3 * (n + m) + nnz;
  printf("flen = %lu\n", flen);

  Sparse<T, I, O>& A = pogs_data->factors;
  A.val = 0;
  A.ind = 0;
  A.ptr = 0;
  A.nnz = nnz;

  hipError_t err = hipMalloc(&A.val, 2 * flen * sizeof(T));
  if (err == hipSuccess)
    err = hipMemset(A.val, 0, 2 * flen * sizeof(T));
  if (err != hipSuccess) {
    hipFree(A.val);
    return 1;
  }

  err = hipMalloc(&A.ind, 2 * nnz * sizeof(I));
  if (err == hipSuccess)
    err = hipMemset(A.ind, 0, 2 * nnz * sizeof(I));
  if (err != hipSuccess) {
    hipFree(A.ind);
    hipFree(A.val);
    return 1;
  }

  err = hipMalloc(&A.ptr, (m + n + 2) * sizeof(I));
  if (err == hipSuccess)
    err = hipMemset(A.ptr, 0, (m + n + 2) * sizeof(I));
  if (err != hipSuccess) {
    hipFree(A.ptr);
    hipFree(A.ind);
    hipFree(A.val);
    return 1;
  }

  return 0;
}

template <typename T, typename I, POGS_ORD O>
void FreeSparseFactors(PogsData<T, Sparse<T, I,O> > *pogs_data) {
  Sparse<T, I, O> &A = pogs_data->factors;
  hipFree(A.ptr);
  hipFree(A.ind);
  hipFree(A.val);

  A.val = 0;
  A.ind = 0;
  A.ptr = 0;
}


// Declarations.
template int Pogs<double, Sparse<double, int, COL> >
    (PogsData<double, Sparse<double, int, COL> > *);
template int Pogs<double, Sparse<double, int, ROW> >
    (PogsData<double, Sparse<double, int, ROW> > *);
template int Pogs<float, Sparse<float, int, COL> >
    (PogsData<float, Sparse<float, int, COL> > *);
template int Pogs<float, Sparse<float, int, ROW> >
    (PogsData<float, Sparse<float, int, ROW> > *);

template int AllocSparseFactors<double, int, ROW>
    (PogsData<double, Sparse<double, int, ROW> > *);
template int AllocSparseFactors<double, int, COL>
    (PogsData<double, Sparse<double, int, COL> > *);
template int AllocSparseFactors<float, int, ROW>
    (PogsData<float, Sparse<float, int, ROW> > *);
template int AllocSparseFactors<float, int, COL>
    (PogsData<float, Sparse<float, int, COL> > *);

template void FreeSparseFactors<double, int, ROW>
    (PogsData<double, Sparse<double, int, ROW> > *);
template void FreeSparseFactors<double, int, COL>
    (PogsData<double, Sparse<double, int, COL> > *);
template void FreeSparseFactors<float, int, ROW>
    (PogsData<float, Sparse<float, int, ROW> > *);
template void FreeSparseFactors<float, int, COL>
    (PogsData<float, Sparse<float, int, COL> > *);

